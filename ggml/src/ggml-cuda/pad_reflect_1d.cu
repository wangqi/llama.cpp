#include "hip/hip_runtime.h"
#include "pad_reflect_1d.cuh"

static __global__ void pad_reflect_1d_kernel_f32(
    const void * __restrict__ src0,
    void * __restrict__ dst,
    const int64_t ne0,
    const int64_t ne00,
    const int64_t ne01,
    const int64_t ne02,
    const int64_t ne03,
    const int64_t nb00,
    const int64_t nb01,
    const int64_t nb02,
    const int64_t nb03,
    const int64_t nb0,
    const int64_t nb1,
    const int64_t nb2,
    const int64_t nb3,
    const int p0,
    const int p1) {

    const int64_t i3 = blockIdx.z;
    const int64_t i2 = blockIdx.y;
    const int64_t i1 = blockIdx.x;

    if (i1 >= ne01 || i2 >= ne02 || i3 >= ne03) {
        return;
    }

    const char * src0_ptr = (const char *)src0 + i3*nb03 + i2*nb02 + i1*nb01;
    char * dst_ptr = (char *)dst + i3*nb3 + i2*nb2 + i1*nb1;

    for (int64_t i0 = threadIdx.x; i0 < ne0; i0 += blockDim.x) {
        float value;

        if (i0 < p0) {
            // Left padding - reflect
            value = *(const float *)(src0_ptr + (p0 - i0) * nb00);
        } else if (i0 < ne0 - p1) {
            // Middle - copy
            value = *(const float *)(src0_ptr + (i0 - p0) * nb00);
        } else {
            // Right padding - reflect
            int64_t src_idx = (ne0 - p1 - p0) - (p1 + 1 - (ne0 - i0)) - 1;
            value = *(const float *)(src0_ptr + src_idx * nb00);
        }

        *(float *)(dst_ptr + i0 * nb0) = value;
    }
}

void ggml_cuda_op_pad_reflect_1d(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type == GGML_TYPE_F32);

    const int32_t * opts = (const int32_t *) dst->op_params;
    const int p0 = opts[0];
    const int p1 = opts[1];

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne0 = dst->ne[0];

    GGML_ASSERT(ne0 == ne00 + p0 + p1);

    const dim3 block_dims(CUDA_PAD_REFLECT_1D_BLOCK_SIZE, 1, 1);
    const dim3 grid_dims(ne01, ne02, ne03);

    pad_reflect_1d_kernel_f32<<<grid_dims, block_dims, 0, stream>>>(
        src0->data, dst->data,
        ne0, ne00, ne01, ne02, ne03,
        src0->nb[0], src0->nb[1], src0->nb[2], src0->nb[3],
        dst->nb[0], dst->nb[1], dst->nb[2], dst->nb[3],
        p0, p1
    );
}
